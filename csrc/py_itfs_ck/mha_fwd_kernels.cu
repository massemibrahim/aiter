// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.

#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include "py_itfs_common.h"
#include "mha_common.h"

#include "mha_fwd.h"

namespace aiter {
namespace torch_itfs {
fmha_fwd_args get_ck_fmha_fwd_args(bool has_lse,
                                   bool has_dropout_randval,
                                   const mask_info &mask,
                                   // sizes
                                   const int b,
                                   const int seqlen_q,
                                   const int seqlen_k,
                                   const int h,
                                   const int h_k,
                                   const int d,
                                   const int d_v,
                                   // device pointers
                                   const at::Tensor q,
                                   const at::Tensor k,
                                   const at::Tensor v,
                                   std::optional<const at::Tensor> &bias_,
                                   std::optional<const at::Tensor> &alibi_slopes_,
                                   at::Tensor out,
                                   at::Tensor softmax_lse,
                                   at::Tensor dropout_randval,
                                   float softmax_scale,
                                   float p_dropout,
                                   std::pair<uint64_t*, uint64_t*> drop_seed_offset)
{
    // q: (batch_size, seqlen_q, nheads, d)
    // k: (batch_size, seqlen_k, nheads_k, d)
    // v: (batch_size, seqlen_k, nheads_k, d_v)
    // o: (batch_size, seqlen_q, nheads, d_v)

    // bias:(seqlen_q, seqlen_k)
    // alibi_slopes:(batch_size, nheads) or (nhead)
    // lse: (batch_size, nheads, seqlen_q)
    // randval: (batch_size, nheads, seqlen_q, seqlen_k)

    ck_tile::index_t stride_q = q.stride(1);
    ck_tile::index_t stride_k = k.stride(1);
    ck_tile::index_t stride_v = v.stride(1);
    ck_tile::index_t stride_o = out.stride(1);
    ck_tile::index_t stride_randval = has_dropout_randval ? dropout_randval.stride(2) : 0;

    ck_tile::index_t nhead_stride_q = q.stride(2);
    ck_tile::index_t nhead_stride_k = k.stride(2);
    ck_tile::index_t nhead_stride_v = v.stride(2);
    ck_tile::index_t nhead_stride_o = out.stride(2);
    ck_tile::index_t nhead_stride_lse = has_lse ? softmax_lse.stride(1) : 0;
    ck_tile::index_t nhead_stride_randval = has_dropout_randval ? dropout_randval.stride(1) : 0;

    ck_tile::index_t batch_stride_q = q.stride(0);
    ck_tile::index_t batch_stride_k = k.stride(0);
    ck_tile::index_t batch_stride_v = v.stride(0);
    ck_tile::index_t batch_stride_o = out.stride(0);

    ck_tile::index_t batch_stride_lse = has_lse ? softmax_lse.stride(0) : 0;
    ck_tile::index_t batch_stride_randval = has_dropout_randval ? dropout_randval.stride(0) : 0;

    void *bias_ptr = nullptr;
    ck_tile::index_t stride_bias = 0;

    if (bias_.has_value()) {
        auto bias = bias_.value();
        CHECK_DEVICE(bias);
        TORCH_CHECK(bias.stride(-1) == 1, "bias tensor must have contiguous last dimension");
        TORCH_CHECK(bias.sizes() == torch::IntArrayRef({seqlen_q, seqlen_k}), "bias shape should be [sq, sk]");
        bias_ptr = bias.data_ptr();
        stride_bias = bias.stride(0);
    }
    else if (alibi_slopes_.has_value()) {
        auto alibi_slopes = alibi_slopes_.value();
        CHECK_DEVICE(alibi_slopes);
        TORCH_CHECK(alibi_slopes.stride(-1) == 1, "ALiBi slopes tensor must have contiguous last dimension");
        TORCH_CHECK(alibi_slopes.sizes() == torch::IntArrayRef({h}) || alibi_slopes.sizes() == torch::IntArrayRef({b, h}));
        bias_ptr = alibi_slopes.data_ptr();
        stride_bias = alibi_slopes.dim() == 2 ? alibi_slopes.stride(0) : 0;
    }

    return fmha_fwd_args{q.data_ptr(),
                         k.data_ptr(),
                         v.data_ptr(),
                         bias_ptr,
                         has_dropout_randval ? dropout_randval.data_ptr() : nullptr,
                         has_lse ? softmax_lse.data_ptr() : nullptr,
                         out.data_ptr(),
                         nullptr, // seqstart_q
                         nullptr, // seqstart_k
                         nullptr,
                         seqlen_q,
                         seqlen_k,
                         b,
                         seqlen_q,      // max_seqlen_q
                         d,             // hdim_q
                         d_v,           // hdim_v
                         h,             // nhead
                         h_k,           // nhead_k
                         softmax_scale, // scale_s
                         1,             // scale_p
                         1,             // scale_o
                         stride_q,
                         stride_k,
                         stride_v,
                         stride_bias,
                         stride_randval,
                         stride_o,
                         nhead_stride_q,
                         nhead_stride_k,
                         nhead_stride_v,
                         0, // nhead_stride_bias
                         nhead_stride_randval,
                         nhead_stride_lse,
                         nhead_stride_o,
                         batch_stride_q,
                         batch_stride_k,
                         batch_stride_v,
                         0, // batch_stride_bias
                         batch_stride_randval,
                         batch_stride_lse,
                         batch_stride_o,
                         mask.left,
                         mask.right,
                         static_cast<ck_tile::index_t>(mask.type),
                         p_dropout,
                         has_dropout_randval,
                         drop_seed_offset};
}

std::vector<at::Tensor>
mha_fwd(at::Tensor &q, // [b, sq, hq, d]
        const at::Tensor &k, // [b, sk, hk, d]
        const at::Tensor &v, // [b, sk, hk, d_v]
        float p_dropout,
        float softmax_scale,
        bool is_causal,
        int window_size_left,
        int window_size_right,
        bool return_softmax_lse,
        bool return_dropout_randval,
        std::optional<at::Tensor> out_,          // [b, sq, hq, d_v]
        std::optional<const at::Tensor> bias_,   // [sq, sk]
        std::optional<const at::Tensor> alibi_slopes_, // [hq] or [b, hq]
        std::optional<at::Generator> gen_)
{
    std::cout << "*** MAI - Enter mha_fwd from aiter/csrc/py_itfs_ck/mha_fwd_kernels.cu" << std::endl;

    auto q_dtype = q.dtype();
    TORCH_CHECK(q_dtype == torch::kFloat16 || q_dtype == torch::kBFloat16,
                "FlashAttention only support fp16 and bf16 data type");

    TORCH_CHECK(k.dtype() == q_dtype, "query and key must have the same dtype");
    TORCH_CHECK(v.dtype() == q_dtype, "query and value must have the same dtype");

    std::string q_dtype_str = q_dtype == torch::kFloat16 ? "fp16" : "bf16";

    CHECK_DEVICE(q); CHECK_DEVICE(k); CHECK_DEVICE(v);

    TORCH_CHECK(q.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(k.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(v.stride(-1) == 1, "Input tensor must have contiguous last dimension");

    const auto sizes = q.sizes();

    const int batch_size = sizes[0];
    int seqlen_q = sizes[1];
    int num_heads = sizes[2];
    const int head_size_q = sizes[3];
    const int head_size_v = v.sizes()[3];
    const int seqlen_k = k.size(1);
    const int num_heads_k = k.size(2);
    TORCH_CHECK(batch_size > 0, "batch size must be positive");
    TORCH_CHECK(head_size_q <= 256, "CK only supports head dimension at most 256");
    TORCH_CHECK(head_size_v <= 256, "CK only supports head dimension at most 256");
    TORCH_CHECK(head_size_q % 8 == 0, "query, key, value, and out_ must have a head_size_q that is a multiple of 8");
    TORCH_CHECK(head_size_v % 8 == 0, "query, key, value, and out_ must have a head_size_q that is a multiple of 8");
    TORCH_CHECK(num_heads % num_heads_k == 0, "Number of heads in key/value must divide number of heads in query");

    if (window_size_left >= seqlen_k) { window_size_left = -1; }
    if (window_size_right >= seqlen_k) { window_size_right = -1; }

    // causal=true is the same as causal=false in this case
    if (seqlen_q == 1 && !alibi_slopes_.has_value()) { is_causal = false; }

    mask_info mask;
    if (is_causal) {
        // Causal is the special case where window_size_right == 0 and window_size_left < 0.
        window_size_right = 0;
        std::string mask_identify = "b:" + std::to_string(window_size_left) + "," + "0";
        mask = mask_info::decode(mask_identify, seqlen_q, seqlen_k); // casual
    }
    else if (window_size_left == -1 && window_size_right == -1) {
        mask = mask_info::decode("0", seqlen_q, seqlen_k); // no mask
    }
    else {
        // Local is the more general case where window_size_right >= 0 or window_size_left >= 0.
        std::string mask_identify = "b:" + std::to_string(window_size_left) + "," + std::to_string(window_size_right);
        mask = mask_info::decode(mask_identify, seqlen_q, seqlen_k); // local
    }

    TORCH_CHECK(!(bias_.has_value() && alibi_slopes_.has_value()), "cannot apply bias and alibi at the same time");
    bias_enum bias_type = bias_.has_value() ? bias_enum::elementwise_bias :
        alibi_slopes_.has_value() ? bias_type = bias_enum::alibi : bias_enum::no_bias;

    // Faster to transpose q from (b, 1, (nheads_kv ngroups), d) to (b, ngroups, nheads_kv, d) in this case
    // H/t Daniel Haziza
    const int seqlenq_ngroups_swapped = seqlen_q == 1 && num_heads > num_heads_k &&
        window_size_left < 0 && window_size_right < 0 && p_dropout == 0.f && head_size_q % 8 == 0 &&
        !alibi_slopes_.has_value() && !bias_.has_value();
    const int ngroups = num_heads / num_heads_k;
    if (seqlenq_ngroups_swapped) {
        q = q.reshape({batch_size, num_heads_k, ngroups, head_size_q}).transpose(1, 2);
        seqlen_q = ngroups;
        num_heads = num_heads_k;
    }

    CHECK_SHAPE(q, batch_size, seqlen_q, num_heads, head_size_q);
    CHECK_SHAPE(k, batch_size, seqlen_k, num_heads_k, head_size_q);
    CHECK_SHAPE(v, batch_size, seqlen_k, num_heads_k, head_size_v);

    auto opts = q.options();
    at::Tensor out;
    if (out_.has_value()) {
        out = out_.value();
        TORCH_CHECK(out.dtype() == q_dtype, "Output must have the same dtype as inputs");
        CHECK_DEVICE(out);
        TORCH_CHECK(out.stride(-1) == 1, "Output tensor must have contiguous last dimension");
        CHECK_SHAPE(out, batch_size, sizes[1], sizes[2], head_size_v);
        if (seqlenq_ngroups_swapped) {
            out = out.reshape({batch_size, num_heads_k, ngroups, head_size_v}).transpose(1, 2);
        }
    }
    else {
        out = torch::empty({batch_size, seqlen_q, num_heads, head_size_v}, opts.dtype(q_dtype));
    }

    // Otherwise the kernel will be launched from cuda:0 device
    at::cuda::CUDAGuard device_guard{q.device()};

    bool has_lse = return_softmax_lse;
    bool has_dropout = p_dropout > 0.0f;

    at::Tensor softmax_lse;
    if (return_softmax_lse) {
        softmax_lse = torch::empty({batch_size, num_heads, seqlen_q}, opts.dtype(torch::kFloat32));
    }
    else {
        softmax_lse = torch::empty({ 0 }, opts.dtype(torch::kFloat32));
    }

    at::Tensor p;
    if (return_dropout_randval) {
        TORCH_CHECK(has_dropout, "return_dropout_randval require p_dropout > 0");
        p = torch::empty({batch_size, num_heads, seqlen_q, seqlen_k}, opts.dtype(torch::kUInt8));
    }
    else {
        p = torch::empty({ 0 }, opts);
    }

    int64_t counter_offset = batch_size * num_heads * ck_tile::get_warp_size();
    auto rng_state = torch::empty({2}, opts.dtype(torch::kInt64));
    auto rng_state_ptr = reinterpret_cast<uint64_t*>(rng_state.data_ptr());

    if (p_dropout > 0.0)  {
        auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
            gen_, at::cuda::detail::getDefaultCUDAGenerator());
        // See Note [Acquire lock when using random generators]
        std::lock_guard<std::mutex> lock(gen->mutex_);
        auto philox_args = gen->philox_cuda_state(counter_offset);
        hipLaunchKernelGGL(
            aiter::ParsePhiloxCudaState, dim3(1), dim3(64), 0, 0, philox_args, rng_state_ptr);
    }

    std::cout << "*** MAI - Check seqlen_k > 0" << std::endl;
    if (seqlen_k > 0) {
        auto drop_seed_offset = std::make_pair(rng_state_ptr, rng_state_ptr + 1);
        auto stream = at::cuda::getCurrentHIPStream().stream();
        ck_tile::stream_config stream_config{stream};

        std::cout << "*** MAI - Call get_ck_fmha_fwd_args" << std::endl;
        auto args =
            get_ck_fmha_fwd_args(
                has_lse,
                return_dropout_randval,
                mask,
                batch_size,
                seqlen_q,
                seqlen_k,
                num_heads,
                num_heads_k,
                head_size_q,
                head_size_v,
                q,
                k,
                v,
                bias_,
                alibi_slopes_,
                out,
                softmax_lse,
                p,
                softmax_scale,
                p_dropout,
                drop_seed_offset);

        std::cout << "*** MAI - Call aiter::mha_fwd" << std::endl;
        std::cout << "*** MAI - This calls the mha_fwd function generated in aiter/csrc/cpp_itfs/mha_fwd_generate.py which calls fmha_fwd from composable_kernel/example/ck_tile/01_fmha/fmha_fwd.hpp" << std::endl;
        std::cout << "*** MAI - fmha_fwd is auto generated using aiter/csrc/cpp_itfs/mha_fwd_generate.py to a file named fmha_fwd_api.cpp" << std::endl;
        std::cout << "*** MAI - I found this file (after it is generated) in aiter/jit/build/mha_fwd_bf16_nbias_nmask_lse_ndropout/blob/fmha_fwd_api.cpp" << std::endl;
        std::cout << "*** MAI - fmha_fwd in fmha_fwd_api.cpp calls fmha_fwd_<trait_>" << std::endl;
        std::cout << "*** MAI - fmha_fwd_<trait_> is defined in aiter/3rdparty/composable_kernel/example/ck_tile/01_fmha/fmha_fwd.hpp (and in aiter/jit/build/<module_name>/build/include/fmha_fwd.hpp after copying/build)" << std::endl;
        std::cout << "*** MAI - fmha_fwd_<trait_> is generated using aiter/3rdparty/composable_kernel/example/ck_tile/01_fmha/codegen/ops/fmha_fwd.py, and calls ck_tile::launch_kernel" << std::endl;
        // MAI - I think this call the mha_fwd function generated in aiter/csrc/cpp_itfs/mha_fwd_generate.py which calls fmha_fwd from composable_kernel/example/ck_tile/01_fmha/fmha_fwd.hpp
        float t = aiter::mha_fwd(args,
                                 stream_config,
                                 mask,
                                 q_dtype_str,
                                 false, // is_group_mode
                                 bias_type,
                                 has_lse);
        TORCH_CHECK(t >= 0, "invalid argument for fmha_fwd");
    }
    else {
        // If seqlen_k == 0, then we have an empty tensor. We need to set the output to 0.
        out.zero_();
        softmax_lse.fill_(std::numeric_limits<float>::infinity());
    }

    if (seqlenq_ngroups_swapped) {
        out = out.transpose(1, 2).reshape({batch_size, 1, num_heads_k * seqlen_q, head_size_q});
        q = q.transpose(1, 2).reshape({batch_size, 1, num_heads_k * seqlen_q, head_size_q});
        if (has_lse) {
            softmax_lse = softmax_lse.reshape({batch_size, num_heads_k * seqlen_q, 1});
        }
    }
    return {out, softmax_lse, p, rng_state};
}

} // namespace torch_itfs
} // namespace aiter
